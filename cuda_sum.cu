#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void cuda_sum_kernel(float *a, float *b, float *c, size_t size)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size) {
		return;
	}

	c[idx] = a[idx] + b[idx];
}

extern "C" {
void cuda_sum(float *a, float *b, float *c, size_t size)
{
	float *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a, size * sizeof(float));
	hipMalloc((void **)&d_b, size * sizeof(float));
	hipMalloc((void **)&d_c, size * sizeof(float));

	hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

	cuda_sum_kernel <<< ceil(size / 256.0), 256 >>> (d_a, d_b, d_c, size);

	hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
}

// attempt to interpolate linear memory
__global__
void cuda_texture_interpolate(hipTextureObject_t tex,
							  float *x,
							  float *y,
							  int n)
   {
   	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx > n) return;

  	y[idx] = tex1D<float>(tex, x[idx]);

/*
  if (count < 1) { count = 1; }
  float h = (stop-start)/((float)count);
  float x = start;
  float y;
  for (int i = 0; i != count; i++) {
	y = tex1D<float>(tex,x);
	printf("x: %4g ; y: %4g\n",x,y);
	x = x + h;
  }
  y = tex1D<float>(tex,x);
  printf("x: %4g ; y: %4g\n",x,y);*/
}

extern "C" {
void cuda_interp1D(float *a, float *c, float *grid, size_t size, size_t num_interps)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, size);
	hipMemcpyToArray(cuArray, 0, 0, a, size*sizeof(float), hipMemcpyHostToDevice);
	
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 1;
	//texDesc.normalizedCoords = 0;

	hipResourceViewDesc resViewDesc;
	memset(&resViewDesc, 0, sizeof(resViewDesc));
	resViewDesc.format = hipResViewFormatFloat1;
	resViewDesc.width = size;

	// create texture object
	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, &resViewDesc);

	// make c and interp grid dev pointers
	float *d_c;
	float *d_grid;
	hipMalloc((void **)&d_c, num_interps * sizeof(float));
	hipMalloc((void **)&d_grid, num_interps * sizeof(float));
	hipMemcpy(d_grid, grid, num_interps * sizeof(float), hipMemcpyHostToDevice);

	cuda_texture_interpolate<<<ceil(num_interps / 256), 256>>>(tex, d_grid, d_c, num_interps);

	 // copy c back to host
	hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

	// clean up
	hipFree(d_grid);
	hipFree(d_c);

	hipDestroyTextureObject(tex);
	hipFreeArray(cuArray);
}

}