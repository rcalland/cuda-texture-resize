#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>


/*extern "C" {
void cuda_float4_array(float4 *a) {
	printf(a[0]);
}

}*/

/////////////////////////////

__global__ void cuda_sum_kernel(float *a, float *b, float *c, size_t size)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size) {
		return;
	}

	printf("f");
	c[idx] = a[idx] + b[idx];
}

extern "C" {
void cuda_sum(float *a, float *b, float *c, size_t size)
{
	float *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a, size * sizeof(float));
	hipMalloc((void **)&d_b, size * sizeof(float));
	hipMalloc((void **)&d_c, size * sizeof(float));

	hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

	cuda_sum_kernel <<< ceil(size / 256.0), 256 >>> (d_a, d_b, d_c, size);

	hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
}

// attempt to interpolate linear memory
__global__
void cuda_texture_interpolate(hipTextureObject_t tex,
							  //float *x,
							  //float4 *y,
							  float4 *out,
							  size_t n)
   {
   	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx >= n) return;

	printf("%i %i\n", idx, n);
  	//out[idx].x = out[idx].y = out[idx].z = out[idx].w = 0.25; //tex1D<float4>(tex, 0.5);
}

__global__
void hello_world(hipTextureObject_t tex, float4 *value) {
	printf("\nhi kernel\n");
	value[0].x = 15.0;
	value[0] = tex2D<float4>(tex, 5., 0.5);
	printf("float4 %f %f %f %f", value[0].x, value[0].y, value[0].z, value[0].w);
}

__device__ size_t flatten_2d_index(size_t x, size_t y, size_t w) {
	// write me
	return (y * w) + x;
}

__global__
void resize_kernel(hipTextureObject_t tex, float4 *output, size_t outw, size_t outh) {
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	size_t idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx >= outw || idy >= outh) return;

	// interpolation coordinates (assumes normalized texture coords!!)
	float int_x = idx * (1.0f / float(outw-1));
	float int_y = idy * (1.0f / float(outh-1));

	size_t out_idx = flatten_2d_index(idx, idy, outw);
	//float4 tmp; 
	//tmp.x = tmp.y = tmp.z = tmp.w = 0.5;
	output[out_idx] = tex2D<float4>(tex, int_x, int_y);
}

extern "C" {
void cuda_resize(float4 *image, float4 *new_image, size_t sizew, size_t sizeh, size_t neww, size_t newh)
{
	//size_t n = sizew * sizeh;
	//printf("starting %f", a[0].x);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, sizew, sizeh);
	hipMemcpyToArray(cuArray, 0, 0, image, sizew*sizeh, hipMemcpyHostToDevice);
	
	//printf("making res desc");
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	//printf("makign tex desc");
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.addressMode[1]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 1;
	//texDesc.normalizedCoords = 0;

	/*hipResourceViewDesc resViewDesc;
	memset(&resViewDesc, 0, sizeof(resViewDesc));
	resViewDesc.format = hipResViewFormatFloat1;
	resViewDesc.width = sizew;
	*/

	// create texture object
	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

	// make c and interp grid dev pointers
	float4 *d_new_image;
	//float *d_grid;
	checkCudaErrors(hipMalloc((void **)&d_new_image, neww * newh * sizeof(float4)));
	//hipMalloc((void **)&d_grid, num_interps * sizeof(float4));
	//hipMemcpy(d_grid, grid, num_interps * sizeof(float4), hipMemcpyHostToDevice);

	//printf("launching kernel");
	//int num_interps = 1024;
	//cuda_texture_interpolate<<<ceil(num_interps / 256), 256>>>(tex, d_out, num_interps);



	/*
	float4 *dev;
	hipMalloc(&dev, sizeof(float4));
	hello_world<<<1,1>>>(tex, dev);

	float4 host;
	hipMemcpy(&host, dev, sizeof(float4), hipMemcpyDeviceToHost);
	hipFree(dev);1d index to 2d
	printf("%f\n", host.x);
	*/
	//hipFree(d_out);

	dim3 blocksize(8, 8);
	dim3 gridsize(neww / blocksize.x, newh / blocksize.y);

	resize_kernel<<<gridsize, blocksize>>>(tex, d_new_image, neww, newh);

	// copy c back to host
	hipMemcpy(new_image, d_new_image, neww * newh * sizeof(float4), hipMemcpyDeviceToHost);

	// clean up
	hipFree(d_new_image);

	hipDestroyTextureObject(tex);
	hipFreeArray(cuArray);
}

}
